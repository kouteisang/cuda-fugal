#include <iostream>
#include <hip/hip_runtime.h>

#include <random>
#include <stdio.h>
#include <float.h>

using namespace std;

#define N 37
#define SIZE 32

void init(float *h_k, float *h_u, float *h_v){

    std::default_random_engine generator;
    std::uniform_real_distribution<float> distribution(0.0, 1.0);


    for(int i = 0; i < N*N; i ++){
        // float number = distribution(generator);
        h_k[i] = i;
    }
    
    for(int i = 0; i < N; i ++){
        h_u[i] = 1.0f/N;
        h_v[i] = 1.0f/N;
    }

    return ;
}

__device__ __forceinline__
void atomicMaxFloat(float *addr, float val){
    atomicMax((int*)addr, __float_as_int(val));
} 

__global__ void sinkhorn_log_cuda(float *d_k, float *add, float *res){
    
    int row = blockIdx.x;
    int col = threadIdx.x;
    int tid = threadIdx.x;
    
    float t_max = -FLT_MAX;
    float sum = 0;
    // shared_max store the max value for each row
    __shared__ float shared_max[32];
    __shared__ float shared_sum[32];

    // use local memory for eac threads
    for(int i = col; i < N; i += blockDim.x){
        int idx = row * N + i;
        t_max = fmaxf(t_max, d_k[idx] + logf(add[i]));
    }

    shared_max[tid] = t_max;
    __syncthreads();

   for(int i = blockDim.x; i > 0; i /= 2){
        if(tid + i < N){
            shared_max[tid] = fmaxf(shared_max[tid], shared_max[tid+i]);
        }
        __syncthreads();
   }
//    shared_max[0] store the maximum for each row;
   for(int i = col; i < N; i += blockDim.x){
        int idx = row * N + i;
        sum += expf(d_k[idx] + logf(add[i]) - shared_max[0]); 
   }

   shared_sum[tid] = sum;
   __syncthreads();

   for(int i = blockDim.x; i > 0; i /= 2){
        if(tid + i < N){
            shared_sum[tid] += shared_sum[tid+i];
        }
        __syncthreads(); 
    }

    if(tid == 0){
        res[row] = logf(1) - (logf(shared_sum[0]) + shared_max[0]);
    }

}

int main(){

    size_t bytes = sizeof(float) * N * N;

    float *h_k, *h_u, *h_v, *h_row_max;
    float *d_k, *d_u, *d_v, *d_row_max;

    // for h_cost
    h_k = (float*)malloc(bytes);
    h_u = (float*)malloc(sizeof(float) * N);
    h_v = (float*)malloc(sizeof(float) * N);
    h_row_max = (float*)malloc(sizeof(float) * N);
    init(h_k, h_u, h_v);

    // cuda memeory allocation for GPU
    hipMalloc(&d_k, bytes);
    hipMalloc(&d_u, sizeof(float) * N);
    hipMalloc(&d_v, sizeof(float) * N);
    hipMalloc(&d_row_max, sizeof(float) * N);
     

    //copy memeory from host to device
    hipMemcpy(d_k, h_k, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_u, h_u, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_v, h_v, sizeof(float) * N, hipMemcpyHostToDevice);

    int BLOCK_SIZE = min(SIZE, 1024);
    int GRID_SIZE = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

    dim3 threads(SIZE);
    dim3 grid(N);

    // calculate the sinkhorn log cuda
    sinkhorn_log_cuda<<<grid, threads>>>(d_k, d_v, d_row_max);
    
    // calculate the maximum value for each row
    hipMemcpy(h_row_max, d_row_max, sizeof(float) * N, hipMemcpyDeviceToHost);

    for(int i = 0; i < N; i ++){
        std::cout << "id = " << i << " max value = " << h_row_max[ i ] << std::endl;
    }

    float ground_truth = 0;
    for(int i = 0; i < N; i ++){
        ground_truth += (i+1-N);
    }

    return 0;
}