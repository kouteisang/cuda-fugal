#include <iostream>
#include <hip/hip_runtime.h>

#include <random>
#include <stdio.h>
#include <float.h>

using namespace std;

#define N 37
#define SIZE 32

void init(float *h_k, float *h_u, float *h_v){

    std::default_random_engine generator;
    std::uniform_real_distribution<float> distribution(0.0, 1.0);


    for(int i = 0; i < N*N; i ++){
        // float number = distribution(generator);
        h_k[i] = i;
    }
    
    for(int i = 0; i < N; i ++){
        h_u[i] = 1.0f/N;
        h_v[i] = 1.0f/N;
    }

    return ;
}

__device__ __forceinline__
void atomicMaxFloat(float *addr, float val){
    atomicMax((int*)addr, __float_as_int(val));
} 

__global__ void matrix_transpose_cuda(float *d_t_k, float *d_k){
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    
    if(row < N && col < N){
        d_t_k[col*N + row] = d_k[row*N + col];
    }
}

__global__ void sinkhorn_log_cuda(float *d_k, float *add, float *res){
    
    int row = blockIdx.x;
    int col = threadIdx.x;
    int tid = threadIdx.x;
    
    float t_max = -FLT_MAX;
    float sum = 0;
    // shared_max store the max value for each row
    __shared__ float shared_max[32];
    __shared__ float shared_sum[32];

    // use local memory for eac threads
    for(int i = col; i < N; i += blockDim.x){
        int idx = row * N + i;
        t_max = fmaxf(t_max, d_k[idx] + logf(add[i]));
    }

    shared_max[tid] = t_max;
    __syncthreads();

   for(int i = blockDim.x; i > 0; i /= 2){
        if(tid + i < N){
            shared_max[tid] = fmaxf(shared_max[tid], shared_max[tid+i]);
        }
        __syncthreads();
   }
//    shared_max[0] store the maximum for each row;
   for(int i = col; i < N; i += blockDim.x){
        int idx = row * N + i;
        sum += expf(d_k[idx] + logf(add[i]) - shared_max[0]); 
   }

   shared_sum[tid] = sum;
   __syncthreads();

   for(int i = blockDim.x; i > 0; i /= 2){
        if(tid + i < N){
            shared_sum[tid] += shared_sum[tid+i];
        }
        __syncthreads(); 
    }

    if(tid == 0){
        res[row] = logf(1) - (logf(shared_sum[0]) + shared_max[0]);
    }

}

int main(){

    size_t bytes = sizeof(float) * N * N;

    float *h_k, *h_u, *h_v, *h_row_max, *h_k_t;
    float *d_k, *d_u, *d_v, *d_row_max, *d_t_k;

    // for h_cost
    h_k = (float*)malloc(bytes);
    h_k_t = (float*)malloc(bytes); 
    h_u = (float*)malloc(sizeof(float) * N);
    h_v = (float*)malloc(sizeof(float) * N);
    h_row_max = (float*)malloc(sizeof(float) * N);
    init(h_k, h_u, h_v);

    // cuda memeory allocation for GPU
    hipMalloc(&d_k, bytes);
    hipMalloc(&d_t_k, bytes);
    hipMalloc(&d_u, sizeof(float) * N);
    hipMalloc(&d_v, sizeof(float) * N);
    hipMalloc(&d_row_max, sizeof(float) * N);
     

    //copy memeory from host to device
    hipMemcpy(d_k, h_k, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_u, h_u, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_v, h_v, sizeof(float) * N, hipMemcpyHostToDevice);


    matrix_transpose_cuda<<<dim3((N + SIZE - 1) / SIZE, (N + SIZE - 1) / SIZE), dim3(SIZE, SIZE)>>>(d_t_k, d_k);

    int BLOCK_SIZE = min(SIZE, 1024);
    int GRID_SIZE = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

    dim3 threads(SIZE);
    dim3 grid(N);

    // calculate the sinkhorn log cuda
    sinkhorn_log_cuda<<<grid, threads>>>(d_t_k, d_u, d_v);
    sinkhorn_log_cuda<<<grid, threads>>>(d_k, d_v, d_u);

    // cudaMemcpy(h_k_t, d_t_k, bytes, cudaMemcpyDeviceToHost);
    // calculate the maximum value for each row
    // cudaMemcpy(h_row_max, d_row_max, sizeof(float) * N, cudaMemcpyDeviceToHost);

    // for(int i = 0; i < N; i ++){
    //     std::cout << "id = " << i << " max value = " << h_row_max[ i ] << std::endl;
    // }
    
    // for(int i = 0; i < N; i ++){
    //     for(int j = 0; j < N; j ++){
    //         std::cout<< h_k_t[i*N+j] << " ";
    //     }
    //     std::cout << std::endl;
    // }
    

    return 0;
}